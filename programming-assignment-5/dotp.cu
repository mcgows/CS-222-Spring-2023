#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>

__global__ void dotp(float* u, float* v, float* partialSum);

const int N = 256 * 256 * 256;
const int threadsPerBlock = 256;
const int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

int getDeviceInfo() {
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount(&count);
    printf("there are %d device(s)\n", count);
    for (int i = 0; i < count; ++i) {
        hipGetDeviceProperties(&prop, i);
        printf("name is %s\n", prop.name);
        printf("major.minor is %d.%d\n", prop.major, prop.minor);
        printf("multiProcessorCount is %d\n", prop.multiProcessorCount);
        printf("warpSize is %d\n", prop.warpSize);
        printf("maxThreadsPerBlock is %d\n", prop.maxThreadsPerBlock);
        printf("maxThreadsDim is (%d, %d, %d)\n", prop.maxThreadsDim[0],
            prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("maxGridSize is (%d, %d, %d)\n", prop.maxGridSize[0],
            prop.maxGridSize[1], prop.maxGridSize[2]);
        if (prop.deviceOverlap)
            printf("device overlap is enabled\n");
        else
            printf("device overlap is NOT enabled\n");
    }

    return 0;
}

float dotp_cpu(float* u, float* v) {
	float temp = 0.0;
	for (int i = 0; i < N; i++) {
		temp += u[i] * v[i];
	}
	return temp;
}

__global__
void dotp(float* u, float* v, float* partialSum) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	float temp = 0.0;

	while (tid < N) {
		temp += u[tid] * v[tid];
		tid += stride;
	}
	// set the cache values
	cache[cacheIndex] = temp;

	// synchronize threads in this block
	__syncthreads();

	// for reductions, threadsPerBlock must be a power of 2
	// because of the following code
	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
		partialSum[blockIdx.x] = cache[0];
}

int main() {
	float *U, *V, w, * partialSum;
	float *dev_U, *dev_V, *dev_Z;

	hipEvent_t startNoMem, startMem, stop, cpuStart, cpuStop;
	hipEventCreate(&startNoMem);
	hipEventCreate(&startMem);
	hipEventCreate(&stop);
	hipEventCreate(&cpuStart);
	hipEventCreate(&cpuStop);

	// allocate memory on the cpu side
	U = (float *) malloc (N * sizeof(float));
	V = (float *) malloc (N * sizeof(float));
	partialSum = (float *) malloc (numBlocks * sizeof(float));

	// fill in the host mempory with data
	// NOTE : this methodology came from stack overflow as I needed to convert from drand48() to windows specific commands.
	srand(time(NULL));
	for (int i = 0; i < N; i++) {
		U[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
		V[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
	}

	hipEventRecord(startMem, 0);
	// allocate the memory on the gpu
	hipMalloc((void**)&dev_U, N * sizeof(float));
	hipMalloc((void**)&dev_V, N * sizeof(float));
	hipMalloc((void**)&dev_Z, numBlocks * sizeof(float));

	// copy the arrays 'u' and 'v' to the gpu
	hipMemcpy(dev_U, U, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_V, V, N * sizeof(float), hipMemcpyHostToDevice);


	hipEventRecord(startNoMem, 0);
	dotp <<<numBlocks, threadsPerBlock>>> (dev_U, dev_V, dev_Z);
	hipDeviceSynchronize(); // wait for GPU threads to complete; again, not necessary but good pratice
	hipEventRecord(stop, 0);

	// copy the array 'dev_Z' back from the gpu to the cpu into partialSum
	hipMemcpy(partialSum, dev_Z, numBlocks * sizeof(float), hipMemcpyDeviceToHost);


	// finish up on the cpu side
	w = 0;
	for (int i = 0; i < numBlocks; i++) {
		w += partialSum[i];
	}


	hipEventRecord(cpuStart, 0);
	float cpuDotpSum = dotp_cpu(U, V);
	hipEventRecord(cpuStop, 0);

	float elapsedTimeNoMem, elapsedTimeMem, elapsedTimeCpu;
	hipEventElapsedTime(&elapsedTimeMem, startMem, stop);
	hipEventElapsedTime(&elapsedTimeNoMem, startNoMem, stop);
	hipEventElapsedTime(&elapsedTimeCpu, cpuStart, cpuStop);


	printf("Relative Error [ GPU vs. CPU] rounded to ten thousandths of a percent : %.3g%%\n", (abs(w) - abs(cpuDotpSum)) / abs(w));
	printf("GPU Execution time [ Memory Operations Counted ] : %f ms\n", elapsedTimeMem);
	printf("GPU Execution time [ No Memory Operations ] : %f ms\n", elapsedTimeNoMem);
	printf("CPU Execution time : %g ms\n", elapsedTimeCpu);


	// free memory on the gpu side
	hipFree(dev_U);
	hipFree(dev_V);
	hipFree(dev_Z);

	// free memory on the cpu side
	free(U);
	free(V);
	free(partialSum);

    return 0;
}